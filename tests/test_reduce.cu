#include "hip/hip_runtime.h"
#include <moderngpu/kernel_reduce.hxx>
#include <moderngpu/memory.hxx>
#include <numeric> // std:accumulate

using namespace mgpu;
// template<typename type_t>
// struct quad{
//   int left_element;
//   int left_count;

//   int current_element;
//   int current_count;
  
//   int best_element;
//   int best_count;

//   int right_element;
//   int right_count;
  
// };
// template<typename type_t>
int main(int argc, char** argv) {

  standard_context_t context;

<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
  typedef launch_params_t<32 * 6, 11> launch_t;
=======
  typedef launch_params_t<32*6, 4> launch_t;
>>>>>>> 842d8b8... Pushing from daisy
=======
  typedef launch_params_t<32*6, 10> launch_t;
>>>>>>> 35d91ae7346f5df65ed2b208093b07f118ec01fd

  for(int count = 1280; count < 1281; count += count / 100) {
    mem_t<int> input = // fill_random(0, 100, count, false, context);
<<<<<<< HEAD
    const int* input_data = input.data();
      // fill(2, count, context);
      fill_random(1, 1000, count, true,context);
=======
      fill(2, count, context);
>>>>>>> 35d91ae7346f5df65ed2b208093b07f118ec01fd
    int* input_data = input.data();

    

    mem_t<quad> reduction(1, context);

    // printf("Is there an error? %d", from_mem(reduction).at(0).current_count);
    // return 0;
    reduce<launch_t>(input_data, count, reduction.data(), perform_t<int>(), perform_t<quad>(), 
      context);
    context.synchronize();
    std::vector<quad> result1 = from_mem(reduction);
    printf("reduce:  %d\t%d\t%d\t%d\t%d\n", result1[0].best_count, result1[0].best_element, result1[0].left_count, result1[0].right_count, result1[0].current_count);
    // // transform_reduce()
=======
  typedef launch_params_t<32*8, 5> launch_t;

  for(int count = 1280; count < 1281; count += count / 100) {
    mem_t<int> input = // fill_random(0, 100, count, false, context);
      // fill(2, count, context);
      fill_random(7, 100, count, true,context);
     int* input_data = input.data();

    

    mem_t<quad> reduction(1, context);

    // printf("Is there an error? %d", from_mem(reduction).at(0).current_count);
    // return 0;
    
>>>>>>> seg_reduce
    // // construct a lambda that returns input_data[index].
    // auto f = [=]MGPU_DEVICE(int index) { return input_data[index]; };
    // //transform_reduce(f, count, reduction.data(), plus_t<int>(), context);
    // std::vector<int> result2 = from_mem(reduction);

    // // host reduce using std::accumulate.
<<<<<<< HEAD
    // std::vector<int> input_host = from_mem(input);
    // int ref = std::accumulate(input_host.begin(), input_host.end(), 0);
=======
    std::vector<int> input_host = from_mem(input);
    int counter = 1;
        int max = 0;
        int mode = input_host.at(0);
        for (int pass = 0; pass < input_host.size() - 1; pass++)
        {
           if ( input_host.at(pass) ==input_host.at(pass+1) )
           {
              counter++;
              if ( counter > max )
              {
                  max = counter;
                  mode = input_host.at(pass);
              }
           } else
              counter = 1; // reset counter.
        }
    printf("Mode is %d and value is %d", max, mode);
    reduce<launch_t>(input_data, count, reduction.data(), perform_t<int>(), perform_t<quad>(), 
      context);
    context.synchronize();
    std::vector<quad> result1 = from_mem(reduction);
    printf("reduce:  %d\t%d\t%d\t%d\n", result1[0].best_count, result1[0].best_element, result1[0].left_count, result1[0].right_count);
    // // transform_reduce()
    // for(int i=0; i < input_host.size(); i++)
    // printf("%d\n", input_host.at(i));    // int ref = std::accumulate(input_host.begin(), input_host.end(), 0);
>>>>>>> seg_reduce

    // if(result1[0] != ref || result2[0] != ref) {
    //   printf("reduce:           %d\n", result1[0]);
    //   printf("transform_reduce: %d\n", result2[0]);
    //   printf("std::accumulate:  %d\n", ref);
    //   printf("ERROR AT COUNT = %d\n", count);
    //   exit(1);
    // } else
    //   printf("Reduction for count %d success\n", count);
  }
  return 0; 

}
