#include "hip/hip_runtime.h"
#include <moderngpu/kernel_mergesort.hxx>

using namespace mgpu;

void printMode(int a[], int n) 
{ 
	    // The output array b[] will 
	    // have sorted array 
	    int b[n]; 
	      
	        // variable to store max of 
	        // input array which will 
	        // to have size of count array 
	        int max = 100000;
		  
		    // auxiliary(count) array to 
		    // store count. Initialize 
		    // count array as 0. Size 
		    // of count array will be 
		    // equal to (max + 1). 
		    int t = max + 1; 
		    int count[t]; 
			    for (int i = 0; i < t; i++) count[i] = 0; 
			      
			        // Store count of each element 
			        // of input array 
			        for (int i = 0; i < n; i++){  
					if(a[i]>100000) printf("%d", a[i]);
					count[a[i]]++; 
				}
				    // mode is the index with maximum count 
				    int mode = 0; 
				        int k = count[0]; 
					    for (int i = 1; i < t; i++) { 
						            if (count[i] > k) { 
								                k = count[i]; 
										            mode = i; 
											            } 
							        } 
					     printf("%d,", mode);

} 

int main(int argc, char** argv) {
  standard_context_t context;

  // Loop from 1K to 100M.
  for(int count = 2000; count <= 1000000; count += count / 10) {
    for(int it = 1; it <= 5; ++it) {

      mem_t<int> data = fill_random(0, 100000, count, false, context);

      mergesort(data.data(), count, less_t<int>(), context);

      std::vector<int> ref = from_mem(data);
    int arr[ref.size()];
    std::copy(ref.begin(), ref.end(), arr);    
      //std::sort(ref.begin(), ref.end());
    printMode(arr, count);  
    //std::vector<int> sorted = from_mem(data);
	printf("%d\n", from_mem(data).at(0));
      

    }
  }

  return 0;
}

